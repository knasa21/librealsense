
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-force-flattening-filter.cuh"
#include "../../../include/librealsense2/rsutil.h"
#include "../../cuda/rscuda_utils.cuh"
#include <cuda_runtime.h>
#include "device_launch_parameters.h"
#include <vector>
#include "thrust/device_vector.h"

using namespace librealsense;
using namespace rscuda;

#define CUDA_THREAD_SIZE 32;



// CUDA�֐��G���[�`�F�b�N�}�N��
#define CHECK(call)															\
{																			\
	const cudaError_t error = call;											\
	if ( error != cudaSuccess )												\
	{																		\
		printf("Error: %s:%d, ", __FILE__, __LINE__ );						\
		printf("code:%d, reason: %s\n", error, cudaGetErrorString(error));	\
		exit(1);															\
	}																		\
}	

__constant__ float coef_P1[3][3] = {
	{0.105f, 0.324f, 0.105f, },
	{0.324f, 1.f, 0.324f, },
	{0.105f, 0.324f, 0.105f, },
};

__constant__ float coef_P5[11 * 11];

// �J�[�l���T�C�Y���Ƃ̋����̕��U(�W���΍��Ђ̓��)
__constant__ float _sqr_space_sigma_array[5] = { 0.444f, 1.12f, 2.811f, 6.829f, 15.122 };

// �W���΍��̌v�Z
__device__ float calc_dispersion( const float* vals, const int size );
//{
//	float sum = 0;
//	// ��0�l�̐�
//	float count = 0;
//
//	for ( int i = 0; i < size; ++i )
//	{
//		float val = vals[i];
//		sum += val;
//		if ( val != 0 )
//		{
//			++count;
//		}
//	}
//
//	// 0�����Ȃ��Ƃ�0��Ԃ� 
//	if ( count == 0 )
//	{
//		return 0;
//	}
//
//	float mean = sum / count;
//	sum = 0;
//
//	for ( int i = 0; i < size; ++i )
//	{
//		float val = vals[i];
//		if ( val != 0 )
//		{
//			sum += std::powf( mean - val, 2 );
//		}
//	}
//
//	return sum / count;
//}


// lab�l�̋���
__device__ float lab_distance( const float* r_lab, const float* l_lab );
//{
//	return
//		std::powf( r_lab[0] - l_lab[0], 2 )
//		+ std::powf( r_lab[1] - l_lab[1], 2 )
//		+ std::powf( r_lab[2] - l_lab[2], 2 );
//}


__device__ float gamma_expanded( const float u );
//{
//	return u > 0.04045 ? pow( (u + 0.055) / 1.055, 2.4 ) : (u / 12.92);
//}


__device__ void convert_rgb8_to_xyz( const uint8_t* rgb, float& x, float& y, float& z );
//{
//	// RGB�̒l��0.0~1.0��
//	float r = rgb[0] / 255.0f;
//	float g = rgb[1] / 255.0f;
//	float b = rgb[2] / 255.0f;
//
//	// �K���}�␳�̏���
//	r = gamma_expanded( r );
//	g = gamma_expanded( g );
//	b = gamma_expanded( b );
//
//	// XYZ�ɕϊ�
//	x = 0.4124 * r + 0.3576 * g + 0.1805 * b;
//	y = 0.2126 * r + 0.7152 * g + 0.0722 * b;
//	z = 0.0193 * r + 0.1192 * g + 0.9505 * b;
//
//	x *= 100;
//	y *= 100;
//	z *= 100;
//}

__global__ void kernel_convert_rgb8_to_lab( const uint8_t* rgb_in, float* lab_out, const uint16_t width, const uint16_t height );
//{
//	int x = blockIdx.x * blockDim.x + threadIdx.x;
//	int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//	// �����x6,500K�̒����F��z�肵���W������D65��XYZ�l
//	float xn = 95.05f, yn = 100.0f, zn = 108.91f;
//
//	// xyz�̒l�̕␳�@pow( 24/116, 3 ) = 0.008856, 1 / ( 3*pow(24/116) ) = 7.787037
//	auto f = []( float val )
//	{
//		return val > 0.008856 ? std::powf( val, 1.0f / 3.0f ) : (7.787037 * val) + 0.137931;
//	};
//
//	int idx = y * width + x;
//
//	if ( x < width && y < height )
//	{
//		// RGB,L*a*b���ꂼ��̕ҏW�ʒu�̃|�C���^
//		const uint8_t* p_rgb = rgb_in + idx * 3;
//		float* p_lab = lab_out + idx * 3;
//
//		// sRGB����CIE XYZ�֕ϊ�
//		float x, y, z;
//		convert_rgb8_to_xyz( p_rgb, x, y, z );
//
//		float fx = f( x / xn );
//		float fy = f( y / yn );
//		float fz = f( z / zn );
//
//		p_lab[0] = 116 * fy - 16;
//		p_lab[1] = 500 * (fx - fy);
//		p_lab[2] = 200 * (fy - fz);
//
//	}
//
//}

void __global__ kernel_hole_filter_with_color( uint16_t* depth_out, const uint16_t* depth_in, const float* lab_in, const int kernel_w, const int width, const int height );
//{
//	const int x = blockIdx.x * blockDim.x + threadIdx.x;
//	const int y = blockIdx.y * blockDim.y + threadIdx.y;
//	const int idx = y * width + x;
//
//	if ( x < width - kernel_w && y < height - kernel_w && x > kernel_w && y > kernel_w )
//	{
//		// �����͈�
//		const int image_kernel_size = kernel_w * 2 + 1;
//
//		int size = kernel_w * 2 + 1;
//		//float* lab_dists = new float[size*size];
//		//float* sqrt_lab_dists = new float[size*size];
//
//		// ��ɐF������z��ɑ�� 
//		/*for ( int n = -kernel_w; n <= kernel_w; ++n )
//		{
//			for ( int m = -kernel_w; m <= kernel_w; ++m )
//			{
//				int source = (y + n) * width + (x + m);
//				float lab_dist = lab_distance( &lab_in[idx * 3], &lab_in[source * 3] );
//
//				int um = m + kernel_w;
//				int un = n + kernel_w;
//				lab_dists[un*size + um] = lab_dist;
//				sqrt_lab_dists[un*size + um] = lab_dist * lab_dist * 3;
//			}
//		}*/
//
//		// �F�����̃V�O�}�l�͕W���΍�
//		const float sqr_color_sigma = 5;// calc_dispersion( lab_dists, size * size );
//
//		//delete[] lab_dists;
//		//delete[] sqrt_lab_dists;
//
//		// �W���΍�0�͑ł��؂�
//		if ( sqr_color_sigma == 0 )
//		{
//			return;
//		}
//
//		float denominator = 0;
//		float numerator = 0;
//
//		int black_counter = 0;
//		// �l�v�Z
//		for ( int n = -kernel_w; n <= kernel_w; ++n )
//		{
//			for ( int m = -kernel_w; m <= kernel_w; ++m )
//			{
//				const int source = (y + n) * width + (x + m);
//				const int k = (n + kernel_w) * size + (m + kernel_w);
//
//				// �����͊܂߂Ȃ�
//				if ( n == 0 && m == 0 )
//				{
//					continue;
//				}
//
//				// �[�x0�͖�������
//				if ( depth_in[source] == 0 )
//				{
//					continue;
//				}
//
//				float lab_dist = lab_distance( &lab_in[idx * 3], &lab_in[source * 3] );
//				if ( lab_dist > 5 ) continue;
//				const float P = coef_P5[k];
//				const float N = std::exp( -lab_dist / (2 * sqr_color_sigma) );
//
//				numerator += depth_in[source] * P * N;
//				denominator += P * N;
//			}
//		}
//
//		if ( numerator != 0 && denominator != 0 )
//		{
//			depth_out[idx] = (uint16_t)(numerator / denominator);
//		}
//
//
//		/*depth_out[idx] =
//			coef_P[0][0] * depth_in[(y - 1)*width + x - 1] +
//			coef_P[0][1] * depth_in[(y - 1)*width + x] +
//			coef_P[0][2] * depth_in[(y - 1)*width + x + 1] +
//			coef_P[1][0] * depth_in[(y)*width + x - 1] +
//			coef_P[1][1] * depth_in[(y)*width + x] +
//			coef_P[1][2] * depth_in[(y)*width + x + 1] +
//			coef_P[2][0] * depth_in[(y + 1)*width + x - 1] +
//			coef_P[2][1] * depth_in[(y + 1)*width + x] +
//			coef_P[2][2] * depth_in[(y + 1)*width + x + 1];*/
//
//	}
//}

void force_flattening_filter_cuda_helper::convert_rgb8_to_lab( const uint8_t* h_rgb_in, float* h_lab_out, const uint16_t width, const uint16_t height )
{
	// �z��̗v�f���ƃo�C�g��
	const int size = width * height * 3;
	const int labBytes = sizeof( float ) * size;
	const int rgbBytes = sizeof( uint8_t ) * size;

	// �f�o�C�X�������̊m�ہA�z�X�g����̓]��
	if ( !_d_rgb_in ) { _d_rgb_in = alloc_dev<uint8_t>( size ); }
	cudaMemcpy( _d_rgb_in.get(), h_rgb_in, rgbBytes, cudaMemcpyHostToDevice );

	// �f�o�C�X�������̊m�ہA������
	if ( !_d_lab_out ) { _d_lab_out = alloc_dev<float>( size ); }
	cudaMemset( _d_lab_out.get(), 0, labBytes );

	// �J�[�l���̃X���b�h���ݒ�
	const int threadSize = CUDA_THREAD_SIZE;
	dim3 threads( threadSize, threadSize );
	dim3 blocks( (width + threadSize - 1) / threadSize, (height + threadSize - 1) / threadSize );

	// �J�[�l���Ăяo��
	kernel_convert_rgb8_to_lab << <blocks, threads >> > (_d_rgb_in.get(), _d_lab_out.get(), width, height);

	// CUDA�����҂�
	cudaDeviceSynchronize();

	// �f�o�C�X����������z�X�g�������֓]��
	cudaMemcpy( h_lab_out, _d_lab_out.get(), labBytes, cudaMemcpyDeviceToHost );
}

void librealsense::force_flattening_filter_cuda_helper::hole_filter_process( uint16_t* h_depth_out, const uint16_t* h_depth_in, const float* h_lab_in, const int kernel_w, const int width, const int height )
{
	// �z��̗v�f���ƃo�C�g��
	const int depthSize = width * height;
	const int depthBytes = sizeof( uint16_t ) * depthSize;
	const int labSize = width * height * 3;
	const int labBytes = sizeof( float ) * labSize;

	// �f�o�C�X�������̊m�ہA�z�X�g����̓]��
	if ( !_d_depth_in ) { _d_depth_in = alloc_dev<uint16_t>( depthSize ); }
	cudaMemcpy( _d_depth_in.get(), h_depth_in, depthBytes, cudaMemcpyHostToDevice );

	// �f�o�C�X�������̊m�ہA�z�X�g����̓]��
	if ( !_d_lab_in ) { _d_lab_in = alloc_dev<float>( labSize ); }
	cudaMemcpy( _d_lab_in.get(), h_lab_in, labBytes, cudaMemcpyHostToDevice );

	// �f�o�C�X�������̊m�ہA������
	if ( !_d_depth_out ) { _d_depth_out = alloc_dev<uint16_t>( depthSize ); }
	//cudaMemset( _d_depth_out.get(), 0, depthBytes );
	cudaMemcpy( _d_depth_out.get(), h_depth_in, depthBytes, cudaMemcpyHostToDevice );

	// P�l�̔z����쐬
	if ( _h_coef_P5[0] == 0 ) { initializeCoef(); }
	int kernel_size = kernel_w * 2 + 1;

	// �J�[�l���̃X���b�h���ݒ�
	const int threadSize = CUDA_THREAD_SIZE;
	dim3 threads( threadSize, threadSize );
	dim3 blocks( (width + threadSize - 1) / threadSize, (height + threadSize - 1) / threadSize );

	const int roop_time = 6;
	for ( int i = 0; i < roop_time; ++i )
	{
		kernel_hole_filter_with_color << <blocks, threads >> > (_d_depth_out.get(), _d_depth_in.get(), _d_lab_in.get(), kernel_w, width, height);

		// CUDA�����҂�
		cudaDeviceSynchronize();

		// ���̓f�[�^�̍X�V
		cudaMemcpy( _d_depth_in.get(), _d_depth_out.get(), depthBytes, cudaMemcpyDeviceToDevice );
	}

	// �J�[�l���̃G���[�`�F�b�N
	CHECK( cudaGetLastError() );

	// �f�o�C�X����������z�X�g�������֓]��
	cudaMemcpy( h_depth_out, _d_depth_out.get(), depthBytes, cudaMemcpyDeviceToHost );
}

// P�l�̌W��(��ԋ���)�̏�����
void librealsense::force_flattening_filter_cuda_helper::initializeCoef()
{
	const int kernel_w = 5;
	const float sqr_space_sigma = 10;
	int size = kernel_w * 2 + 1;
	for ( int n = -kernel_w; n <= kernel_w; ++n )
	{
		for ( int m = -kernel_w; m <= kernel_w; ++m )
		{
			float space_distance = n * n + m * m;
			float P = std::exp( -space_distance / (2.0f * sqr_space_sigma) );
			int um = m + kernel_w;
			int un = n + kernel_w;
			_h_coef_P5[un * size + um] = P;
		}
	}

	// �R���X�^���g�������ւ̃A�b�v���[�h
	for ( int i = 0; i < 11; ++i )
	{
		cudaMemcpyToSymbol( coef_P5, _h_coef_P5, 11 * 11 * sizeof( float ), cudaMemcpyHostToDevice );
	}
}


#endif // RS2_USE_CUDA
