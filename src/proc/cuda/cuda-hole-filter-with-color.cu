
#include <hip/hip_runtime.h>
#ifdef RS2_USE_CUDA

#include "cuda-hole-filter-with-color.cuh"
#include "../../../include/librealsense2/rsutil.h"
#include "../../cuda/rscuda_utils.cuh"
#include <cuda_runtime.h>
#include "device_launch_parameters.h"

using namespace librealsense;
using namespace rscuda;

__device__ float gamma_expanded( const float u )
{
	return u > 0.04045 ? pow( ( u+0.055 )/1.055, 2.4 ) : ( u / 12.92 );
}


__device__ void convert_rgb8_to_xyz( const uint8_t* rgb, float& x, float& y, float& z )
{
	// RGB�̒l��0.0~1.0��
	float r = rgb[0] / 255.0f;
	float g = rgb[1] / 255.0f;
	float b = rgb[2] / 255.0f;

	// �K���}�␳�̏���
	r = gamma_expanded( r );
	g = gamma_expanded( g );
	b = gamma_expanded( b );

	// XYZ�ɕϊ�
	x = 0.4124 * r + 0.3576 * g + 0.1805 * b;
	y = 0.2126 * r + 0.7152 * g + 0.0722 * b;
	z = 0.0193 * r + 0.1192 * g + 0.9505 * b;

	x *= 100;
	y *= 100;
	z *= 100;
}

__global__ void kernel_convert_rgb8_to_lab( const uint8_t* rgb_in, float* lab_out, const uint16_t width, const uint16_t height )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// �����x6,500K�̒����F��z�肵���W������D65��XYZ�l
	float xn = 95.05f, yn = 100.0f, zn = 108.91f;

	// xyz�̒l�̕␳�@pow( 24/116, 3 ) = 0.008856, 1 / ( 3*pow(24/116) ) = 7.787037
	auto f = []( float val )
	{
		return val > 0.008856 ? std::powf( val, 1.0f / 3.0f ) : (7.787037 * val) + 0.137931;
	};

	int idx = y * width + x;
	
	if( idx < width * height )
	{
		// RGB,L*a*b���ꂼ��̕ҏW�ʒu�̃|�C���^
		const uint8_t* p_rgb = rgb_in + idx * 3;
		float* p_lab = lab_out + idx * 3;

		// sRGB����CIE XYZ�֕ϊ�
		float x, y, z;
		convert_rgb8_to_xyz( p_rgb, x, y, z );

		float fx = f( x / xn );
		float fy = f( y / yn );
		float fz = f( z / zn );

		p_lab[0] = 116 * fy - 16;
		p_lab[1] = 500 * ( fx - fy );
		p_lab[2] = 200 * ( fy - fz );

	}

}

void hole_filter_with_color_cuda_helper::convert_rgb8_to_lab( const uint8_t* h_rgb_in, float* h_lab_out, const uint16_t width, const uint16_t height )
{
	int size = width * height;
	if ( !_d_rgb_in ) { _d_rgb_in = alloc_dev<uint8_t>( size * 3 ); }
	cudaMemcpy( _d_rgb_in.get(), h_rgb_in, sizeof( uint8_t ) * size * 3, cudaMemcpyHostToDevice );

	if ( !_d_lab_out ) { _d_lab_out = alloc_dev<float>( size * 3 ); }
	cudaMemset( _d_lab_out.get(), 0, sizeof( float ) * size * 3 );

	const int threadSize = 32;
	dim3 threads( threadSize, threadSize );
	dim3 blocks( (width + threadSize - 1) / threadSize, (height + threadSize - 1) / threadSize );
	
	kernel_convert_rgb8_to_lab << <blocks, threads >> > (_d_rgb_in.get(), _d_lab_out.get(), width, height);

	cudaDeviceSynchronize();

	cudaMemcpy( h_lab_out, _d_lab_out.get(), size * 3 * sizeof(float), cudaMemcpyDeviceToHost );
}


#endif // RS2_USE_CUDA
